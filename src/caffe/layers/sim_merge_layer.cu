#include "hip/hip_runtime.h"

/********************************************************************************
** Copyright(c) 2016 USTC Reserved.
** auth: Xu Shen
** mail: shenxu@mail.ustc.edu.cn
** date: 2016/1/1
** desc: SimMergeLayer(GPU), merge similar feature maps and re-initialize similar
**       weights to learn more independent feature maps
*********************************************************************************/
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/sim_merge_layer.hpp"

namespace caffe{

	template <typename Dtype>
	__global__ void ComputeSim(const int count, const int N, Dtype *sim_data){
		CUDA_KERNEL_LOOP(index, count){
			const int row = index / N;
			const int col = index % N;
			//sim(\vec{a}, \vec{b}) = (\vec{a} \dot \vec{b}) / 
			//(\sqrt(\vec{a} \dot \vec{a}) \times \sqrt(\vec{b} \dot \vec{b})
			const Dtype sqrt_i = sqrt(sim_data[row * N + row]);
			const Dtype sqrt_j = sqrt(sim_data[col * N + col]);
			sim_data[row * N + col] /= (sqrt_i * sqrt_j);
		}
	}

	//TODO: maybe this operation will be very time consuming, we 
	// need to figure out a more efficient way
	template <typename Dtype>
	void SimMergeLayer<Dtype>::update_sim_matrix_gpu(){
		Dtype* weight_data = this->blobs_[0]->mutable_gpu_data();
		//to save memory, put history similarity in data
		//and current similarity in diff
		Dtype* curr_sim_data = this->sim_.mutable_gpu_diff();
		Dtype* his_sim_data = this->sim_.mutable_gpu_data();
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, N_, N_, K_, Dtype(1.),
			weight_data, weight_data, Dtype(0), curr_sim_data);
		const int count = N_ * N_;
		ComputeSim<Dtype><<< CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, N_, curr_sim_data);
		CUDA_POST_KERNEL_CHECK;
		//update history similarity with current similarity
		if (use_history_){
			const Dtype curr_iter = 1 + this->curr_iter_;
			caffe_gpu_axpby(N_ * N_, (Dtype)1. / (Dtype)curr_iter, curr_sim_data,
				(Dtype)this->curr_iter_ / (Dtype)curr_iter, his_sim_data);
		}
		else{
			caffe_copy<Dtype>(N_ * N_, curr_sim_data, his_sim_data);
		}
	}

	template <typename Dtype>
	void SimMergeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		//currently, we have nothing to do
	}

	template <typename Dtype>
	void SimMergeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		this->update_sim_matrix_gpu();
		this->curr_iter_++;
		if (this->curr_iter_ % this->iter_ == 0){
			//reset number of iterations, 
			//so as to reset similarity matrix to all 0s
			this->curr_iter_ = 0;
			// NOTE: I don't think a gpu version can accelerate the computation
			// so I just use the cpu code here
			this->merge_sim_weights_cpu();
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(SimMergeLayer);
}
