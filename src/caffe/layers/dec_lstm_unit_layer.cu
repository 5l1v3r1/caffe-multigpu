#include "hip/hip_runtime.h"
#include <string>
#include <utility>
#include <vector>

#include "caffe/layers/dec_lstm_unit_layer.hpp"

namespace caffe{
	template <typename Dtype>
	__device__ Dtype sigmoid(Dtype x){
		return Dtype(1) / (Dtype(1) + exp(-x));
	}

	template <typename Dtype>
	__device__ Dtype tanh(Dtype x){
		return Dtype(2) * sigmoid(Dtype(2) * x) - Dtype(1);
	}

	template <typename Dtype>
	__device__ Dtype relu(Dtype x){
		return x > 0 ? x : 0;
	}

	template <typename Dtype>
	__global__ void DLSTMActsForward(const int nthreads, const int dim,
		const Dtype* X, Dtype* X_acts){
		CUDA_KERNEL_LOOP(index, nthreads){
			const int x_dim = 4 * dim;
			const int d = index % x_dim;
			if (d < 3 * dim){
				X_acts[index] = sigmoid(X[index]);
			}
			else{
				X_acts[index] = tanh(X[index]);
			}
		}
	}

	template <typename Dtype>
	__global__ void DLSTMUnitForward(const int nthreads, const int dim,
		const Dtype* C_prev, const Dtype* X, Dtype* C, Dtype* H){
		CUDA_KERNEL_LOOP(index, nthreads){
			const int n = index / dim;
			const int d = index % dim;
			const Dtype* X_offset = X + 4 * dim * n;
			const Dtype i = X_offset[d];
			const Dtype f = X_offset[dim + d];
			const Dtype o = X_offset[2 * dim + d];
			const Dtype g = X_offset[3 * dim + d];
			const Dtype c = f * C_prev[d] + i * g;
			C[d] = c;
			const Dtype tanh_c = tanh(c);
			H[d] = o * tanh_c;
		}
	}

	template <typename Dtype>
	void DLSTMUnitLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		const Dtype* C_prev = bottom[0]->gpu_data();
		const Dtype* X = bottom[1]->gpu_data();
		Dtype* X_acts = X_acts_.mutable_gpu_data();
		Dtype* C = top[0]->mutable_gpu_data();
		Dtype* H = top[1]->mutable_gpu_data();
		const int count = bottom[0]->count();
		const int x_count = bottom[1]->count();
		DLSTMActsForward<Dtype> << <CAFFE_GET_BLOCKS(x_count), CAFFE_CUDA_NUM_THREADS >> >(
			x_count, hidden_dim_, X, X_acts);
		CUDA_POST_KERNEL_CHECK;
		DLSTMUnitForward<Dtype> << < CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, hidden_dim_, C_prev, X_acts, C, H);
		CUDA_POST_KERNEL_CHECK;
	}

	template <typename Dtype>
	__global__ void DLSTMUnitBackward(const int nthreads, const int dim,
		const Dtype* C_prev, const Dtype* X, const Dtype* C,
		const Dtype* C_diff, const Dtype* H_diff, Dtype* C_prev_diff, Dtype* X_diff){
		CUDA_KERNEL_LOOP(index, nthreads){
			const int n = index / dim;
			const int d = index % dim;
			const Dtype* X_offset = X + 4 * dim * n;
			const Dtype i = X_offset[d];
			const Dtype f = X_offset[dim + d];
			const Dtype o = X_offset[2 * dim + d];
			const Dtype g = X_offset[3 * dim + d];
			const Dtype c = C[d];
			const Dtype tanh_c = tanh(c);
			Dtype* X_diff_offset = X_diff + 4 * dim * n;
			Dtype* i_diff = X_diff_offset + d;
			Dtype* f_diff = X_diff_offset + dim + d;
			Dtype* o_diff = X_diff_offset + 2 * dim + d;
			Dtype* g_diff = X_diff_offset + 3 * dim + d;
			Dtype* c_prev_diff = C_prev_diff + d;
			Dtype h_diff = H_diff[d];
			Dtype c_diff = C_diff[d];
			//accumulate diff bp from c_t and h_t
			const Dtype c_term_diff = c_diff + h_diff * (1 - tanh_c * tanh_c);
			*c_prev_diff = c_term_diff * f;
			*g_diff = c_term_diff * i;
			*o_diff = tanh_c * h_diff;
			*f_diff = c_term_diff * C_prev[d];
			*i_diff = c_term_diff * g;
		}
	}

	template <typename Dtype>
	__global__ void DLSTMActsBackward(const int nthreads, const int dim,
		const Dtype* X_acts, const Dtype* X_acts_diff, Dtype* X_diff){
		CUDA_KERNEL_LOOP(index, nthreads){
			const int x_dim = 4 * dim;
			const int d = index % x_dim;
			const Dtype X_act = X_acts[index];
			if (d < 3 * dim){
				X_diff[index] = X_acts_diff[index] * X_act * (Dtype(1) - X_act);
			}
			else{
				X_diff[index] = X_acts_diff[index] * (Dtype(1) - X_act * X_act);
			}
		}
	}

	template <typename Dtype>
	void DLSTMUnitLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){
		const Dtype* C_prev = bottom[0]->gpu_data();
		const Dtype* X = bottom[1]->gpu_data();
		Dtype* X_acts = X_acts_.mutable_gpu_data();
		const Dtype* H_diff = top[1]->gpu_diff();
		const Dtype* C_diff = top[0]->gpu_diff();
		const Dtype* C = top[0]->gpu_data();
		Dtype* X_diff = bottom[1]->mutable_gpu_diff();
		Dtype* X_acts_diff = X_acts_.mutable_gpu_diff();
		Dtype* C_prev_diff = bottom[0]->mutable_gpu_diff();
		const int count = top[1]->count();
		const int x_count = bottom[1]->count();
		//why do forward of action of X again?
		DLSTMActsForward<Dtype> << <CAFFE_GET_BLOCKS(x_count), CAFFE_CUDA_NUM_THREADS >> >(
			x_count, hidden_dim_, X, X_acts);
		CUDA_POST_KERNEL_CHECK;
		DLSTMUnitBackward<Dtype> // NOLINT_NEXT_LINE(whitespace/operators)
			<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>> > (
			count, hidden_dim_, C_prev, X, C, C_diff, H_diff, C_prev_diff, X_acts_diff);
		CUDA_POST_KERNEL_CHECK;
		DLSTMActsBackward<Dtype> // NOLINT_NEXT_LINE(whitespace/operators)
			<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>> > (
			x_count, hidden_dim_, X_acts, X_acts_diff, X_diff);
		CUDA_POST_KERNEL_CHECK;
	}

	INSTANTIATE_LAYER_GPU_FUNCS(DLSTMUnitLayer);

} // namespace caffe
