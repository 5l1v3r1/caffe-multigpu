#include "hip/hip_runtime.h"
/********************************************************************************
** Copyright(c) 2016 USTC Reserved.
** auth: Xu Shen
** mail: shenxu@mail.ustc.edu.cn
** date: 2016/04/29
** desc: PatchRankLayer(GPU)
*********************************************************************************/
#include "caffe/layers/patch_rank_layer.hpp"
#include <thrust/sort.h>

namespace caffe{

	/*
	 * nthreads: total number of unit blocks
	 * num_ * channels_ * num_unit_block_ * num_unit_block_
	 */
	template<typename Dtype>
	__global__ void ComputeBlockEnergyL1(const int nthreads, int split_num,
		const int height, const int width, const int unit_block_height,
		const int unit_block_width, const int num_unit_block, const Dtype* bottom_data,
		Dtype* energy_data, Dtype* index_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int bw = index % num_unit_block;
			int bh = (index / num_unit_block) % num_unit_block;
			int c = index /  num_unit_block / num_unit_block;
			Dtype sum = 0;
			for (int h = 0; h < unit_block_height; ++h){
				for (int w = 0; w < unit_block_width; ++w){
					int offset = c * height * width + bh * unit_block_height * width 
						+ bw * unit_block_width;
					sum += abs(bottom_data[offset + h * width + w]);
				}
			}
			// save data of a single block for next level(split_num * split_num)
			// in continuous memory, which makes sorting faster
			int bw_next = bw / split_num;
			int bh_next = bh / split_num;
			int block_size = split_num * split_num;
			int next_block_offset = bh_next * split_num * block_size + 
				bw_next * block_size;
			int next_inner_bw = bw % split_num;
			int next_inner_bh = bh % split_num;
			energy_data[c * num_unit_block * num_unit_block + 
				next_block_offset + next_inner_bh * split_num + 
				next_inner_bw] = sum;
			index_data[c * num_unit_block * num_unit_block + 
				next_block_offset + next_inner_bh * split_num + 
				next_inner_bw] = bh * num_unit_block + bw;
		}
	}

	/*
	 * nthreads: total number of unit blocks
	 * num_ * channels_ * num_unit_block_ * num_unit_block_
	 */
	template<typename Dtype>
	__global__ void ComputeBlockEnergyL2(const int nthreads, int split_num,
		const int height, const int width, const int unit_block_height,
		const int unit_block_width, const int num_unit_block, const Dtype* bottom_data,
		Dtype* energy_data, Dtype* index_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int bw = index % num_unit_block;
			int bh = (index / num_unit_block) % num_unit_block;
			int c = index / num_unit_block/ num_unit_block;
			Dtype sum = 0;
			for (int h = 0; h < unit_block_height; ++h){
				for (int w = 0; w < unit_block_width; ++w){
					int offset = ( c * height + bh * unit_block_height)
						* width + bw * unit_block_width;
					sum += bottom_data[offset + h * width + w] * 
						bottom_data[offset + h * width + w];
				}
			}
			// save data of a single block for next level(split_num * split_num)
			// in continuous memory, which makes sorting faster
			int bw_next = bw / split_num;
			int bh_next = bh / split_num;
			int block_size = split_num * split_num;
			int next_block_offset = bh_next * split_num * block_size + 
				bw_next * block_size;
			int next_inner_bw = bw % split_num;
			int next_inner_bh = bh % split_num;
			energy_data[c * num_unit_block * num_unit_block + 
				next_block_offset + next_inner_bh * num_unit_block + 
				next_inner_bw] = sum;
			index_data[c * num_unit_block * num_unit_block + 
				next_block_offset + next_inner_bh * num_unit_block +  
				next_inner_bw] = bh * num_unit_block + bw;
		}
	}

  /*
   * nthreads = num_ * channels_ * block_num * block_num
   * each thread get energy of blockes of given pyramid level
   * one single block in this level corresponds to split_num * split_num
   * inner blocks in the previous level
   */
	template<typename Dtype>
	__global__ void ComputeLevelEnergy(int nthreads, int split_num,
		int block_num, const Dtype* prev_level_energy,
		Dtype* level_energy, Dtype* level_index){
		CUDA_KERNEL_LOOP(index, nthreads){
			int patch_num = block_num * split_num;
			int bw = index % block_num;
			int bh = (index / block_num) % block_num;
			int c = index / block_num / block_num;
			//offset in energy map of previous level
			int prev_block_size = split_num * split_num;
			int patch_offset = c * patch_num * patch_num + bh * split_num *
				prev_block_size + bw * prev_block_size;
			Dtype sum = 0;
			//sum in "patch" of this level, computed from previous level data
			for (int h = 0; h < split_num; ++h){
				for (int w = 0; w < split_num; ++w){
					sum += prev_level_energy[patch_offset + h * split_num + w];
				}
			}
			// save data of a single block for next level(split_num * split_num)
			// in continuous memory, which makes sorting faster
			int bw_next = bw / split_num;
			int bh_next = bh / split_num;
			int block_size = split_num * split_num;
			int next_block_offset = bh_next * split_num * block_size + 
				bw_next * block_size;
			int next_inner_bw = bw % split_num;
			int next_inner_bh = bh % split_num;
			level_energy[c * block_num * block_num + 
				next_block_offset + next_inner_bh * block_num + 
				next_inner_bw] = sum;
			level_index[c * block_num * block_num + 
				next_block_offset + next_inner_bh * split_num + 
				next_inner_bw] = bh * block_num + bw;
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::GetBlockEnergy_gpu(const vector<Blob<Dtype>*>& bottom){
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* energy_data = block_infos_[0]->mutable_gpu_data();
		Dtype* index_data = block_infos_[0]->mutable_gpu_diff();
		const int width = bottom[0]->width();
		const int height = bottom[0]->height();
		int count = num_ * channels_ * num_unit_block_ * num_unit_block_;
		switch (energy_type_){
		case PatchRankParameter_EnergyType_L1:
			ComputeBlockEnergyL1<Dtype> << <CAFFE_GET_BLOCKS(count), 
				CAFFE_CUDA_NUM_THREADS >> >(count, split_num_,
				height, width, unit_block_height_, unit_block_width_,
				num_unit_block_, bottom_data, energy_data, index_data);
			CUDA_POST_KERNEL_CHECK;
			break;
		case PatchRankParameter_EnergyType_L2:
			ComputeBlockEnergyL2<Dtype> << <CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS >> >(count, split_num_, 
				height, width, unit_block_height_, unit_block_width_,
				num_unit_block_, bottom_data, energy_data, index_data);
			CUDA_POST_KERNEL_CHECK;
			break;
		default:
			LOG(FATAL) << "Unkown energy type.";
		}
		hipStream_t* stream = new hipStream_t[pyramid_height_ - 1];
		//lunch multi-kernel
		for (int i = 0; i < pyramid_height_ - 1; ++i){
			hipStreamCreate(&stream[i]);
		}
//		block_infos_[0]->ToTxt("block_info_0", true);
		for (int p = 1; p < pyramid_height_; ++p){
			int count = block_infos_[p]->count();
			int block_num = block_infos_[p]->width();
			const Dtype* prev_level_energy = block_infos_[p - 1]->gpu_data();
			Dtype* level_energy = block_infos_[p]->mutable_gpu_data();
			Dtype* level_index = block_infos_[p]->mutable_gpu_diff();
			ComputeLevelEnergy<Dtype> << <1, CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS, stream[p - 1] >> >(count, split_num_,
				block_num, prev_level_energy, level_energy, level_index);
			CUDA_POST_KERNEL_CHECK;
//			ostringstream oss;
//			oss << p;
//			block_infos_[p]->ToTxt("block_info_" + oss.str(), true);
		}
		for (int i = 0; i < pyramid_height_ - 1; ++i){
			hipStreamDestroy(stream[i]);
		}
	}

	template<typename Dtype>
	__device__ void swap(Dtype* data, const int i, const int j){
		Dtype tmp = data[i];
		data[i] = data[j];
		data[j] = tmp;
	}

	/*
	* because the vec will be quite small (split_num_ * split_num_)
	* and generally split_num_ will be set to 2 or 3 or 4
	* we can use bubble sort algorithm
	* both sort on values and indexes
	* in descend order
	*/
	template<typename Dtype>
	__device__ void bubble_sort(const int n, Dtype* values, Dtype* indexes){
		for (int i = 0; i < n; ++i){
			bool swapped = false;
			for (int j = 0; j < n - (i + 1); ++j){
				if (values[j] < values[j + 1]){
					swap<Dtype>(values, j, j + 1);
					swap<Dtype>(indexes, j, j + 1);
					swapped = true;
				}
			}
			if (!swapped){ break; }
		}
	}

	/*
	 * nthreads = num_ * channels_ * next_block_num * next_block_num
	 * the rank in level p will determine the offset of level p - 1
	 */
	template<typename Dtype>
	__global__ void SortInBlock(const int nthreads, const int block_num,
		int split_num, Dtype* energy_data, Dtype* index_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int next_block_num = block_num / split_num;
			int bw = index % next_block_num;
			int bh = (index / next_block_num) % next_block_num;
			int c = index / next_block_num/ next_block_num;
			int block_size = split_num * split_num;
			Dtype* sort_data = energy_data + c * block_num * block_num +
				bh * split_num * block_size + bw * block_size;
			Dtype* sort_index = index_data + c * block_num * block_num +
				bh * split_num * block_size + bw * block_size;
			bubble_sort<Dtype>(block_size, sort_data, sort_index);
		}
	}

	/*
	 * nthreads = num_ * channels_ * block_num * block_num
	 */
	template<typename Dtype>
	__global__ void ComputeOffset(int nthreads, int block_num, int split_num,
		int block_pixel_width, int block_pixel_height,
		const Dtype* index_data, Dtype* offset_w_data, Dtype* offset_h_data,
		Dtype* test_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			// we only care about offset in sub-blocks
			int block_size = split_num * split_num;
			int sorted_iw = (index % block_size) % split_num;
			int sorted_ih = (index % block_size) / split_num;
			int id = int(index_data[index]);
			int c = index / block_num / block_num;
			int source_w =  id % block_num;
			int source_h = (id / block_num) % block_num;
			int source_iw = source_w % split_num;
			int source_ih = source_h % split_num;
			int offset_w = (sorted_iw - source_iw) * block_pixel_width;
			int offset_h = (sorted_ih - source_ih) * block_pixel_height;
			offset_w_data[c * block_num * block_num + id] = offset_w;
			offset_h_data[c * block_num * block_num + id] = offset_h;
			test_data[index] = int(index_data[index]);
		}
	}

  /*
   * @brief pass offsets from level p + 1 to p
   * nthreads = num_ * channels_ * block_num * block_num
   * merge next_level offset into current level offset
   */
	template<typename Dtype>
	__global__ void MergeOffset(int nthreads, int block_num, int split_num, 
		const Dtype* next_offset_w, const Dtype* next_offset_h,
		Dtype* curr_offset_w, Dtype* curr_offset_h){
		CUDA_KERNEL_LOOP(index, nthreads){
			int curr_bw = index % block_num;
			int next_bw = curr_bw / split_num;
			int curr_bh = (index / block_num) % block_num;
			int next_bh = curr_bh / split_num;
			int c = index / block_num / block_num;
			int next_block_num = block_num / split_num;
			int next_index = (c * next_block_num + next_bh) * next_block_num + next_bw;
			curr_offset_w[index] += next_offset_w[next_index];
			curr_offset_h[index] += next_offset_h[next_index];
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::SortBlock_gpu(){
		hipStream_t* stream = new hipStream_t[pyramid_height_];
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamCreate(&stream[i]);
		}
		//sort 
		for (int p = 0; p < pyramid_height_; ++p){
			Dtype* energy_data = block_infos_[p]->mutable_gpu_data();
			Dtype* index_data = block_infos_[p]->mutable_gpu_diff();
			int block_num = block_infos_[p]->width();
			int N = num_ * channels_ * (block_num / split_num_) * 
				(block_num / split_num_);
			SortInBlock<Dtype><< < 1, CAFFE_GET_BLOCKS(N), 
				CAFFE_CUDA_NUM_THREADS, stream[p] >> >(
				N, block_num, split_num_, energy_data, index_data);
			CUDA_POST_KERNEL_CHECK;
//			ostringstream oss;
//			oss << p;
//			block_infos_[p]->ToTxt("block_info_sort_" + oss.str(), true);
		}
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamDestroy(stream[i]);
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::ComputeLevelOffset_gpu(){
		hipStream_t* stream = new hipStream_t[pyramid_height_];
		//lunch multi-kernel to compute offset
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamCreate(&stream[i]);
		}
		Dtype* test_data = test_data_.mutable_gpu_data();
		for (int p = 0; p < pyramid_height_; ++p){
			//offset of level p
			int count = block_offsets_[p]->count();
			int block_num = block_offsets_[p]->width();
			int block_pixel_width = num_unit_block_ / block_num * unit_block_width_;
			int block_pixel_height = num_unit_block_ / block_num * unit_block_height_;
			const Dtype* index_data = block_infos_[p]->gpu_diff();
			Dtype* offset_w_data = block_offsets_[p]->mutable_gpu_data();
			Dtype* offset_h_data = block_offsets_[p]->mutable_gpu_diff();
			ComputeOffset<Dtype> << <1, CAFFE_GET_BLOCKS(count),
				CAFFE_CUDA_NUM_THREADS, stream[p]>> >(count, block_num, split_num_,
				block_pixel_width, block_pixel_height, index_data, offset_w_data, 
				offset_h_data, test_data);
			CUDA_POST_KERNEL_CHECK;
			ostringstream oss;
			oss << p;
			block_offsets_[p]->ToTxt("block_offset_" + oss.str(), true);
		}
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamDestroy(stream[i]);
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::MergeOffset_gpu(){
		hipStream_t* stream = new hipStream_t[pyramid_height_];
		//lunch multi-kernel
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamCreate(&stream[i]);
		}
		//backpropagate offsets from level p to level 1
		for (int p = pyramid_height_ - 2; p >= 0; --p){
			//offset_{p} += offset_{p + 1}
			int count = block_offsets_[p]->count();
			int block_num = block_offsets_[p]->width();
			const Dtype* next_offset_w = block_offsets_[p + 1]->gpu_data();
			const Dtype* next_offset_h = block_offsets_[p + 1]->gpu_diff();
			Dtype* curr_offset_w = block_offsets_[p]->mutable_gpu_data();
			Dtype* curr_offset_h = block_offsets_[p]->mutable_gpu_diff();
			MergeOffset<Dtype> << <1, CAFFE_GET_BLOCKS(count), 
				CAFFE_CUDA_NUM_THREADS, stream[p] >> >(
				count, block_num, split_num_, next_offset_w, next_offset_h,
				curr_offset_w, curr_offset_h);
			CUDA_POST_KERNEL_CHECK;
			ostringstream oss;
			oss << p;
			block_offsets_[p]->ToTxt("block_offset_merge_" + oss.str(), true);
		}
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamDestroy(stream[i]);
		}
	}

	/*
	 * nthreads = num_ * channels_ * height * width
	 */
	template<typename Dtype>
	__global__ void PatchRankForward(const int nthreads,
		const int unit_block_height, const int unit_block_width, 
		const int height, const int width, 
		const int num_unit_block, const Dtype* bottom_data, const Dtype* offset_h_data,
		const Dtype* offset_w_data, Dtype* top_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int w = index % width;
			int h = (index / width) % height;
			int c = index / width / height;
			int block_id_h = h / unit_block_height;
			int block_id_w = w / unit_block_width;
			/*
			 * for pixels not in the sorted blocks
			 * we just copy them to the output
			 */
			if (w >= num_unit_block || h >= num_unit_block){
				top_data[index] = bottom_data[index];
			}
			else{
				int offset_h = static_cast<int>(offset_h_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int offset_w = static_cast<int>(offset_w_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int top_w = w + offset_w;
				int top_h = h + offset_h;
				top_data[c * height * width + top_h * width + top_w] =
					bottom_data[index];
			}
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* offset_w_data = block_offsets_[0]->gpu_data();
		const Dtype* offset_h_data = block_offsets_[0]->gpu_diff();
		Dtype* top_data = top[0]->mutable_gpu_data();
		GetBlockEnergy_gpu(bottom);
		SortBlock_gpu();
		ComputeLevelOffset_gpu();
		MergeOffset_gpu();
		const int count = bottom[0]->count();
		const int height = bottom[0]->height();
		const int width = bottom[0]->width();
		PatchRankForward<Dtype> << < CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, unit_block_height_, unit_block_width_, height, 
			width, num_unit_block_,
			bottom_data, offset_h_data, offset_w_data, top_data);
		CUDA_POST_KERNEL_CHECK;
	}

	template<typename Dtype>
	__global__ void PatchRankBackward(const int nthreads,
		const int unit_block_height, const int unit_block_width, 
		const int height, const int width, 
		const int num_unit_block, const Dtype* top_diff, const Dtype* offset_h_data,
		const Dtype* offset_w_data, Dtype* bottom_diff, Dtype* test_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int w = index % width;
			int h = (index / width) % height;
			int c = index / width / height;
			int block_id_h = h / unit_block_height;
			int block_id_w = w / unit_block_width;
			/*
			 * for pixels not in the sorted blocks
			 * we just copy diffs to the bottom
			 */
			if (block_id_h >= num_unit_block || block_id_w >= num_unit_block){
				bottom_diff[index] = top_diff[index];
			}
			else{
				int offset_h = static_cast<int>(offset_h_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int offset_w = static_cast<int>(offset_w_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int top_w = w + offset_w;
				int top_h = h + offset_h;
				bottom_diff[index] = top_diff[c * height * width + top_h * width + top_w];
				test_data[index] = offset_w;
			}
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom){
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		block_offsets_[0]->ToTxt("offset_bak_0", true);
		const Dtype* offset_w_data = block_offsets_[0]->gpu_data();
		const Dtype* offset_h_data = block_offsets_[0]->gpu_diff();
		const Dtype* top_diff = top[0]->gpu_diff();
		const int count = bottom[0]->count();
		const int height = bottom[0]->height();
		const int width = bottom[0]->width();
		caffe_gpu_set<Dtype>(test_data_.count(), Dtype(0), 
			test_data_.mutable_gpu_data());
		test_data_.ToTxt("test_before");
		PatchRankBackward<Dtype> << < CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, unit_block_height_, unit_block_width_,  
			height, width, num_unit_block_,
			top_diff, offset_h_data, offset_w_data, 
			bottom_diff, test_data_.mutable_gpu_data());
		CUDA_POST_KERNEL_CHECK;
		test_data_.ToTxt("test_data");
	}

	INSTANTIATE_LAYER_GPU_FUNCS(PatchRankLayer);

} // namespace caffe
