#include "hip/hip_runtime.h"
/********************************************************************************
** Copyright(c) 2016 USTC Reserved.
** auth: Xu Shen
** mail: shenxu@mail.ustc.edu.cn
** date: 2016/04/29
** desc: PatchRankLayer(GPU)
*********************************************************************************/
#include "caffe/layers/patch_rank_layer.hpp"

namespace caffe{

	/*
	 * nthreads: total number of unit blocks
	 * num_ * channels_ * num_unit_block_ * num_unit_block_
	 */
	template<typename Dtype>
	__global__ void ComputeBlockEnergyL1(const int nthreads,
		const int height, const int width, const int unit_block_height,
		const int unit_block_width, const int num_unit_block, const Dtype* bottom_data,
		Dtype* energy_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int bw = index % num_unit_block;
			int bh = (index / num_unit_block) % num_unit_block;
			int c = index /  num_unit_block / num_unit_block;
			Dtype sum = 0;
			for (int h = 0; h < unit_block_height; ++h){
				for (int w = 0; w < unit_block_width; ++w){
					int offset = c * height * width + bh * unit_block_height * width 
						+ bw * unit_block_width;
					sum += abs(bottom_data[offset + h * width + w]);
				}
			}
			energy_data[(c * num_unit_block + bh) * num_unit_block
				+ bw] = sum;
		}
	}

	/*
	 * nthreads: total number of unit blocks
	 * num_ * channels_ * num_unit_block_ * num_unit_block_
	 */
	template<typename Dtype>
	__global__ void ComputeBlockEnergyL2(const int nthreads,
		const int height, const int width, const int unit_block_height,
		const int unit_block_width, const int num_unit_block, const Dtype* bottom_data,
		Dtype* energy_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int bw = index % num_unit_block;
			int bh = (index / num_unit_block) % num_unit_block;
			int c = index / num_unit_block/ num_unit_block;
			Dtype sum = 0;
			for (int h = 0; h < unit_block_height; ++h){
				for (int w = 0; w < unit_block_width; ++w){
					int offset = ( c * height + bh * unit_block_height)
						* width + bw * unit_block_width;
					sum += bottom_data[offset + h * width + w] * 
						bottom_data[offset + h * width + w];
				}
			}
			energy_data[(c * num_unit_block + bh) * num_unit_block
				+ bw] = sum;
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::GetBlockEnergy_gpu(const vector<Blob<Dtype>*>& bottom){
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* energy_data = block_energies_.mutable_gpu_data();
		const int width = bottom[0]->width();
		const int height = bottom[0]->height();
		const int total_blocks = num_ * channels_ * num_unit_block_ * num_unit_block_;
		switch (energy_type_){
		case PatchRankParameter_EnergyType_L1:
			ComputeBlockEnergyL1<Dtype> << <CAFFE_GET_BLOCKS(total_blocks), CAFFE_CUDA_NUM_THREADS >> >(
				total_blocks, height, width, unit_block_height_, unit_block_width_,
				num_unit_block_, bottom_data, energy_data);
			CUDA_POST_KERNEL_CHECK;
			break;
		case PatchRankParameter_EnergyType_L2:
			ComputeBlockEnergyL2<Dtype> << <CAFFE_GET_BLOCKS(total_blocks), CAFFE_CUDA_NUM_THREADS >> >(
				total_blocks, height, width, unit_block_height_, unit_block_width_,
				num_unit_block_, bottom_data, energy_data);
			CUDA_POST_KERNEL_CHECK;
			break;
		default:
			LOG(FATAL) << "Unkown energy type.";
		}
	}

	template<typename Dtype>
	__device__ void swap(Dtype* data, const int i, const int j){
		Dtype tmp = data[i];
		data[i] = data[j];
		data[j] = tmp;
	}

  /*
   * because the vec will be quite small (split_num_ * split_num_)
   * and generally split_num_ will be set to 2 or 3 or 4
   * we can use bubble sort algorithm
   * both sort on values and indexes
   * in descend order
   */
	template<typename Dtype>
	__device__ void bubble_sort(const int n, Dtype* values, Dtype* indexes){
		for (int i = 0; i < n; ++i){
			bool swapped = false;
			for (int j = 0; j < n - (i + 1); ++j){
				if (values[j] < values[j + 1]){
					swap<Dtype>(values, j, j + 1);
					swap<Dtype>(indexes, j, j + 1);
					swapped = true;
				}
			}
			if (!swapped){ break; }
		}
	}

  /*
   * nthreads = num_ * channels_ * outer_num * outer_num
   * each thread will sort inside each sub-blocks 
   */
	template<typename Dtype>
	__global__ void ComputeBlockOffset(int nthreads, int split_num, 
		int outer_dim, int outer_num, int num_unit_block,
		int unit_block_height, int unit_block_width,
		const Dtype* energy_data, Dtype* offset_h_data, Dtype* offset_w_data,
		Dtype* test_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			const int num_part = split_num * split_num;
			// 16 KB limitation of local memory for each thread
			Dtype* block_energy = (Dtype*)malloc(num_part * sizeof(Dtype));
			Dtype* indexes = (Dtype*)malloc(num_part * sizeof(Dtype));
			int ow = index % outer_num;
			int oh = (index / outer_num) % outer_num;
			int c = index / outer_num / outer_num;
			int inner_dim = outer_dim / split_num;
			test_data[2 * num_part] = Dtype(inner_dim);
			test_data[2 * num_part + 1] = Dtype(outer_dim);
			int ooffset = c * num_unit_block * num_unit_block + oh *
				outer_dim * num_unit_block + ow * outer_dim;
			for (int ih = 0; ih < split_num; ++ih){
				for (int iw = 0; iw < split_num; ++iw){
					Dtype sum = 0;
					int ioffset = ih * inner_dim * num_unit_block + iw * inner_dim;
					for (int h = 0; h < inner_dim; ++h){
						for (int w = 0; w < inner_dim; ++w){
							sum += energy_data[ooffset + ioffset + h * num_unit_block + w];
						}
					}
					block_energy[ih * split_num + iw] = sum;
					indexes[ih * split_num + iw] = Dtype(ih * split_num + iw);
				}
			}
			//sort
			bubble_sort<Dtype>(num_part, block_energy, indexes);
			for (int i = 0; i < num_part; ++i){
				test_data[i] = block_energy[i];
				test_data[i + num_part] = indexes[i];
			}
			//offset
			for (int b = 0; b < num_part; ++b){
				int sorted_bw = b % split_num;
				int sorted_bh = b / split_num;
				int source_bw = int(indexes[b]) % split_num;
				int source_bh = int(indexes[b]) / split_num;
				//pixel offset in bottom feature map
				int offset_h = (sorted_bh - source_bh) * inner_dim * unit_block_height;
				int offset_w = (sorted_bw - source_bw) * inner_dim * unit_block_width;
				if (offset_h == 0 && offset_w == 0){
					continue;
				}
				//update offset of unit blocks
				for (int h = 0; h < inner_dim; ++h){
					for (int w = 0; w < inner_dim; ++w){
						//accumulated accross different pyramid levels
						offset_w_data[ooffset + source_bh * inner_dim * num_unit_block
							+ source_bw * inner_dim + h * num_unit_block + w] += offset_w;
						offset_h_data[ooffset + source_bh * inner_dim * num_unit_block
							+ source_bw * inner_dim + h * num_unit_block + w] += offset_h;
					}
				}
			}
			free(block_energy);
			free(indexes);
			__syncthreads();
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::GetBlockOffset_gpu(){
		const Dtype* energy_data = block_energies_.gpu_data();
		Dtype* offset_h_data = block_offsets_.mutable_gpu_diff();
		Dtype* offset_w_data = block_offsets_.mutable_gpu_data();
		//clear
		caffe_gpu_set<Dtype>(block_offsets_.count(), Dtype(0), offset_h_data);
		caffe_gpu_set<Dtype>(block_offsets_.count(), Dtype(0), offset_w_data);
		Dtype* test_data = block_energies_.mutable_gpu_diff();
		hipStream_t* stream = new hipStream_t[pyramid_height_];
		//lunch multi-kernel
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamCreate(&stream[i]);
		}
		for (int p = 0; p < pyramid_height_; ++p){
			int outer_dim = pow(split_num_, pyramid_height_ - p);
			int outer_num = pow(split_num_, p);
			int nthreads = num_ * channels_ * outer_num * outer_num;
			ComputeBlockOffset<Dtype> << <1, CAFFE_GET_BLOCKS(nthreads), 
				CAFFE_CUDA_NUM_THREADS, stream[p] >> >(
				nthreads, split_num_, outer_dim, outer_num, num_unit_block_,
				unit_block_height_, unit_block_width_, energy_data, offset_h_data,
				offset_w_data, test_data);
			CUDA_POST_KERNEL_CHECK;
		}
		for (int i = 0; i < pyramid_height_; ++i){
			hipStreamDestroy(stream[i]);
		}
		block_offsets_.ToTxt("offset_gpu",true);
		block_energies_.ToTxt("energy_gpu", true);
	}

	/*
	 * nthreads = num_ * channels_ * height * width
	 */
	template<typename Dtype>
	__global__ void PatchRankForward(const int nthreads,
		const int unit_block_height, const int unit_block_width, 
		const int height, const int width, 
		const int num_unit_block, const Dtype* bottom_data, const Dtype* offset_h_data,
		const Dtype* offset_w_data, Dtype* top_data){
		CUDA_KERNEL_LOOP(index, nthreads){
			int w = index % width;
			int h = (index / width) % height;
			int c = index / width / height;
			int block_id_h = h / unit_block_height;
			int block_id_w = w / unit_block_width;
			/*
			 * for pixels not in the sorted blocks
			 * we just copy them to the output
			 */
			if (w == num_unit_block || h == num_unit_block){
				top_data[index] = bottom_data[index];
			}
			else{
				int offset_h = static_cast<int>(offset_h_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int offset_w = static_cast<int>(offset_w_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int top_w = w + offset_w;
				int top_h = h + offset_h;
				top_data[c * height * width + top_h * width + top_w] =
					bottom_data[index];
			}
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* offset_w_data = block_offsets_.gpu_data();
		const Dtype* offset_h_data = block_offsets_.gpu_diff();
		Dtype* top_data = top[0]->mutable_gpu_data();
		GetBlockEnergy_gpu(bottom);
		GetBlockOffset_gpu();
		const int count = bottom[0]->count();
		const int height = bottom[0]->height();
		const int width = bottom[0]->width();
		PatchRankForward<Dtype> << < CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, unit_block_height_, unit_block_width_, height, 
			width, num_unit_block_,
			bottom_data, offset_h_data, offset_w_data, top_data);
		CUDA_POST_KERNEL_CHECK;
	}

	template<typename Dtype>
	__global__ void PatchRankBackward(const int nthreads,
		const int unit_block_height, const int unit_block_width, 
		const int height, const int width, 
		const int num_unit_block, const Dtype* top_diff, const Dtype* offset_h_data,
		const Dtype* offset_w_data, Dtype* bottom_diff){
		CUDA_KERNEL_LOOP(index, nthreads){
			int w = index % width;
			int h = (index / width) % height;
			int c = index / width / height;
			int block_id_h = h / unit_block_height;
			int block_id_w = w / unit_block_width;
			/*
			 * for pixels not in the sorted blocks
			 * we just copy them to the output
			 */
			if (block_id_h == num_unit_block || block_id_w == num_unit_block){
				bottom_diff[index] = top_diff[index];
			}
			else{
				int offset_h = static_cast<int>(offset_h_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int offset_w = static_cast<int>(offset_w_data[c * num_unit_block *
					num_unit_block + block_id_h * num_unit_block + block_id_w]);
				int top_w = w + offset_w;
				int top_h = h + offset_h;
				bottom_diff[index] = top_diff[c * height * width + top_h * width + top_w];
			}
		}
	}

	template<typename Dtype>
	void PatchRankLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom){
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		const Dtype* offset_w_data = block_offsets_.gpu_data();
		const Dtype* offset_h_data = block_offsets_.gpu_diff();
		const Dtype* top_diff = top[0]->gpu_diff();
		const int count = bottom[0]->count();
		const int height = bottom[0]->height();
		const int width = bottom[0]->width();
		caffe_copy<Dtype>(count, top_diff, bottom_diff);
		PatchRankBackward<Dtype> << < CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, unit_block_height_, unit_block_width_,  
			height, width, num_unit_block_,
			top_diff, offset_h_data, offset_w_data, bottom_diff);
		CUDA_POST_KERNEL_CHECK;
	}

	INSTANTIATE_LAYER_GPU_FUNCS(PatchRankLayer);

} // namespace caffe
