#include "hip/hip_runtime.h"
#include <vector>
#include <utility>
#include <math.h>

#include "caffe/layers/semi_hinge_loss_layer.hpp"

namespace caffe{
	template <typename Dtype>
	void SemiHingeLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top){
		const int count = bottom[0]->count();
		const int dim = bottom[0]->count(axis_);
		const int num = count / dim;
		const Dtype* x1_data = bottom[0]->gpu_data();
		const Dtype* x2_data = bottom[1]->gpu_data();
		Dtype* diff_data = diff_->mutable_gpu_data();
		caffe_gpu_sub(count, x1_data, x2_data, diff_data);
		const Dtype* label_data_1 = bottom[2]->cpu_data();
		const Dtype* label_data_2 = bottom[3]->cpu_data();
		// hinge value
		Dtype* dist_data = dist_->mutable_cpu_data();
		// buff for dist
		Dtype* dist_diff = dist_->mutable_cpu_diff();
		Dtype loss(0.);
		for (int n = 0; n < num; ++n){
			const int label_1 = static_cast<int>(label_data_1[n]);
			const int label_2 = static_cast<int>(label_data_2[n]);
			// D(x_1, x_2)
			Dtype dist;
			caffe_gpu_dot(dim, diff_data, diff_data, &dist);
			if (label_1 != ignore_label_ && label_2 != ignore_label_){
				// indicator
				const int ind = label_1 == label_2 ? 1 : -1;
				// supervised hinge loss
				dist_data[n] = std::max(Dtype(0), sup_bias_ - ind * (sup_thre_ - dist));
				loss += dist_data[n];
			}
			else{
				// unsupervised hinge loss
				dist_data[n] = gamma_ * std::max(Dtype(0), unsup_bias_ - abs(unsup_thre_ - dist));
				// used in backpropagation
				dist_diff[n] = dist;
				loss += dist_data[n];
			}
			diff_data += dim;
		}
		top[0]->mutable_cpu_data()[0] = loss / num / Dtype(2);
	}

	template <typename Dtype>
	void SemiHingeLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom){
		const int num = bottom[0]->count(0, axis_);
		const int dim = bottom[0]->count(axis_);
		Dtype* bottom_diff_1 = bottom[0]->mutable_gpu_diff();
		Dtype* bottom_diff_2 = bottom[1]->mutable_gpu_diff();
		const Dtype* label_data_1 = bottom[2]->cpu_data();
		const Dtype* label_data_2 = bottom[3]->cpu_data();
		const Dtype* diff_data = diff_->gpu_data();
		const Dtype* dist_data = dist_->cpu_data();
		const Dtype* dist_diff = dist_->cpu_diff();
		const Dtype loss_weight = top[0]->cpu_diff()[0];
		for (int n = 0; n < num; ++n){
			const int label_1 = static_cast<int>(label_data_1[n]);
			const int label_2 = static_cast<int>(label_data_2[n]);
			if (label_1 != ignore_label_ && label_2 != ignore_label_){
				// indicator 
				const int ind = label_1 == label_2 ? 1 : -1;
				// supervised
				if (dist_data[n] > 0){
					if (propagate_down[0]){
						caffe_gpu_axpby(dim, Dtype(ind * loss_weight / num), diff_data,
							Dtype(0), bottom_diff_1);
					}
					if (propagate_down[1]){
						caffe_gpu_axpby(dim, Dtype(-ind * loss_weight / num), diff_data,
							Dtype(0), bottom_diff_2);
					}
				}
			}
			else{
				// unsupervised
				if (dist_data[n] > 0){
					Dtype gap = dist_diff[n] - unsup_thre_;
					Dtype alpha = gap > 0 ? 1 : (gap < 0 ? -1 : 0);
	 				if (propagate_down[0]){
						caffe_gpu_axpby(dim, Dtype(gamma_ * alpha * loss_weight / num), diff_data,
							Dtype(0), bottom_diff_1);
					}
					if (propagate_down[1]){
						caffe_gpu_axpby(dim, Dtype(-gamma_ * alpha * loss_weight / num), diff_data,
							Dtype(0), bottom_diff_2);
					}
				}
			}
			diff_data += dim;
			bottom_diff_1 += dim;
			bottom_diff_2 += dim;
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(SemiHingeLossLayer);

} // namespace caffe