#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ClipData(const int n, const Dtype lower, const Dtype higher,
	Dtype* data){
	CUDA_KERNEL_LOOP(index, n){
		Dtype value = data[index];
		data[index] = value > higher ? higher : (value < lower ? lower : value);
	}
}

template <typename Dtype>
void DropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count(0, num_axes_);
  Dtype* mask = rand_vec_->mutable_gpu_data();
  if (this->phase_ == TRAIN) {
	  switch (drop_type_){
	  case DropoutParameter_DropType_BERNOULLI:
	  {
	    // Create random numbers
	    caffe_gpu_rng_bernoulli(count, Dtype(1. - threshold_), mask);
	    break;
	  }
	  case DropoutParameter_DropType_GAUSSIAN:
	  {
	   caffe_gpu_rng_gaussian(count, Dtype(mu_), Dtype(sigma_), mask);
	   // clip to be in [0,1]
	   ClipData<Dtype> // NOLINT_NEXT_LINE(whitespace/operators)
		   << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
		   (count, Dtype(0), Dtype(1), mask);
		CUDA_POST_KERNEL_CHECK;
	   break;
	  }
	  case DropoutParameter_DropType_UNIFORM:
	  {
	    caffe_gpu_rng_uniform(count, Dtype(a_), Dtype(b_), mask);
		break;
	  }
	  }
	  if (drop_batch_){
		  Dtype drop = rand_vec_->cpu_data()[0];
		  drop = 1;
		  caffe_copy(top[0]->count(), bottom_data, top_data);
		  caffe_gpu_scal(top[0]->count(), Dtype(scale_ * drop), top_data);
	  }
	  else{
		  vector<Blob<Dtype>*> scale_bottom(2, NULL);
		  scale_bottom[0] = bottom[0];
		  scale_bottom[1] = rand_vec_;
		  const vector<Blob<Dtype>*> scale_top(1, top[0]);
		  scale_layer_->Forward(scale_bottom, scale_top);
		  caffe_gpu_scal(top[0]->count(), scale_, top_data);
	  }
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* top_diff = top[0]->mutable_gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
		if (drop_batch_){
			Dtype drop = rand_vec_->cpu_data()[0];
			drop = 1;
			caffe_gpu_scal(top[0]->count(), Dtype(scale_ * drop), top_diff);
			caffe_copy(top[0]->count(), top_diff, bottom_diff);
		}
		else{
			// scale
			caffe_gpu_scal<Dtype>(top[0]->count(), scale_, top_diff);
			// multiply mask
			vector<Blob<Dtype>*> scale_bottom(2, NULL);
			scale_bottom[0] = bottom[0];
			scale_bottom[1] = rand_vec_;
			const vector<Blob<Dtype>*> scale_top(1, top[0]);
			vector<bool> prop_down(2, true);
			prop_down[1] = false;
			scale_layer_->Backward(scale_top, prop_down, scale_bottom);
		}
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropoutLayer);

}  // namespace caffe
