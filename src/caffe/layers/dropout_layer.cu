#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ClipData(const int n, const Dtype lower, const Dtype higher,
	Dtype* data){
	CUDA_KERNEL_LOOP(index, n){
		Dtype value = data[index];
		data[index] = value > higher ? higher : (value < lower ? lower : value);
	}
}

template <typename Dtype>
void DropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count(0, num_axes_);
  Dtype* mask = rand_vec_->mutable_gpu_data();
  if (this->phase_ == TRAIN) {
	  switch (drop_type_){
	  case DropoutParameter_DropType_BERNOULLI:
	  {
	    // Create random numbers
	    caffe_gpu_rng_bernoulli(count, Dtype(1. - threshold_), mask);
	    break;
	  }
	  case DropoutParameter_DropType_GAUSSIAN:
	  {
	   caffe_gpu_rng_gaussian(count, Dtype(mu_), Dtype(sigma_), mask);
	   const int mask_count = rand_vec_->count();
	   // clip to be in [0,1]
	   ClipData<Dtype> // NOLINT_NEXT_LINE(whitespace/operators)
		   << <CAFFE_GET_BLOCKS(mask_count), CAFFE_CUDA_NUM_THREADS >> >
		   (count, Dtype(0), Dtype(1), mask);
		CUDA_POST_KERNEL_CHECK;
	   break;
	  }
	  case DropoutParameter_DropType_UNIFORM:
	  {
	    caffe_gpu_rng_uniform(count, Dtype(a_), Dtype(b_), mask);
		break;
	  }
	  }
	  vector<Blob<Dtype>*> scale_bottom(2, NULL);
	  scale_bottom[0] = bottom[0];
	  scale_bottom[1] = rand_vec_;
	  const vector<Blob<Dtype>*> scale_top(1, top[0]);
	  scale_layer_->Forward(scale_bottom, scale_top);
	  caffe_scal(top[0]->count(), scale_, top[0]->mutable_cpu_data());
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* top_diff = top[0]->mutable_gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
		// scale
		caffe_gpu_scal<Dtype>(top[0]->count(), scale_, top_diff);
		// multiply mask
		vector<Blob<Dtype>*> scale_bottom(2, NULL);
		scale_bottom[0] = bottom[0];
		scale_bottom[1] = rand_vec_;
		const vector<Blob<Dtype>*> scale_top(1, top[0]);
		vector<bool> prop_down(2, true);
		prop_down[1] = false;
		scale_layer_->Backward(scale_top, prop_down, scale_bottom);
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropoutLayer);

}  // namespace caffe
