#include "hip/hip_runtime.h"
#include "caffe/util/sim_merge.hpp"
#include <thrust/sort.h>
#include <thrust/functional.h>

namespace caffe{

	template <typename Dtype>
	__global__ void ComputeSim(const int count, const int N, Dtype* sim_data){
		CUDA_KERNEL_LOOP(index, count){
			const int row = index / N;
			const int col = index % N;
			//sim(\vec{a}, \vec{b}) = (\vec{a} \dot \vec{b}) / 
			//(\sqrt(\vec{a} \dot \vec{a}) \times \sqrt(\vec{b} \dot \vec{b})
			const Dtype sqrt_i = sqrt(sim_data[row * N + row]);
			const Dtype sqrt_j = sqrt(sim_data[col * N + col]);
			const Dtype denom = sqrt_i * sqrt_j;
			sim_data[row * N + col] /= denom;
		}
	}

	// set diagonal elements to 0.
	template <typename Dtype>
	__global__ void ResetDiag(const int N, Dtype* sim_data){
		CUDA_KERNEL_LOOP(index, N){
			sim_data[index * N + index] = Dtype(0);
		}
	}

	template <typename Dtype>
	void update_sim_matrix_gpu(Blob<Dtype>* weight, 
		Blob<Dtype>* sim, const int axis){
		//dim_0 * dim_1 * ... * dim_{axis_-1} is the number of output
		const int N = weight->count(0, axis);
		//dim_{axis} * dim_{axis+1} * ... is the number of weights for a single output
		const int K = weight->count(axis);
		CHECK_GT(K, 1) << "similarity can only be computed between vectors";
		// N x N
		const vector<int> sim_shape(2, N);
		sim->Reshape(sim_shape);
		Dtype* weight_data = weight->mutable_gpu_data();
		//to save memory, put history similarity in data
		//and current similarity in diff
		Dtype* sim_data = sim->mutable_gpu_data();
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, N, N, K, Dtype(1.),
			weight_data, weight_data, Dtype(0), sim_data);
		const int count = N * N;
		ComputeSim<Dtype> // NOLINT_NEXT_LINE(whitespace/operators)
			<< < CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
			(count, N, sim_data);
		CUDA_POST_KERNEL_CHECK;
		ResetDiag<Dtype> // NOLINT_NEXT_LINE(whitespace/operators)
			<< < CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS >> >
			(N, sim_data);
		CUDA_POST_KERNEL_CHECK;
	}

	template void update_sim_matrix_gpu<double>(Blob<double>* weight, 
		Blob<double>* sim, const int axis);
	template void update_sim_matrix_gpu<float>(Blob<float>* weight, 
		Blob<float>* sim, const int axis);

	template <typename Dtype>
	void merge_sim_weights_gpu(Blob<Dtype>* weight, Blob<Dtype>* sim,
		const Dtype prop, Filler<Dtype>* filler, const int axis, string name,
		const bool hard){
		// get similarity matrix
		//dim_0 * dim_1 * ... * dim_{axis_-1} is the number of output
		const int N = weight->count(0, axis);
		//dim_{axis} * dim_{axis+1} * ... is the number of weights for a single output
		const int K = weight->count(axis);
		CHECK_GT(K, 1) << "similarity can only be computed between vectors";
		update_sim_matrix_gpu(weight, sim, axis);
		Dtype* weight_data = weight->mutable_gpu_data();
		Dtype* sim_data = sim->mutable_gpu_data();
		Dtype* sim_temp = sim->mutable_gpu_diff();
		Dtype sim_th;
		//get similarity threshold
		const Dtype* sim_data_cpu = sim->cpu_data();
		if (hard){
			// hard: use fixed threshold
			sim_th = prop;
		}
		else{
			// soft: use adaptive threshold
			const int nth = prop * N * N;
			// here the value in sim_data will be changed, so we need to save history
			// similarities in diff for backup
			caffe_copy<Dtype>(sim->count(), sim_data, sim_temp);
			// not working?
			//thrust::sort(sim_temp, sim_temp + sim->count(), thrust::greater<Dtype>());
			Dtype* sim_temp_cpu = sim->mutable_cpu_diff();
			std::nth_element(sim_temp_cpu, sim_temp_cpu + nth,
				sim_temp_cpu + N * N, std::greater<Dtype>());
			sim_th = sim_temp_cpu[nth];
		}
		std::set<int> merged_pos_index;
		LOG(INFO) << "merge positive correlated weights:";
		for (int i = 0; i < N; ++i){
			if (merged_pos_index.count(i)){
				continue;
			}
			for (int j = i + 1; j < N; ++j){
				if (merged_pos_index.count(j)){
					continue;
				}
				const Dtype sim_ij = sim_data_cpu[i * N + j];
				if (sim_ij > sim_th){
					// NOTE: other options: 
					//   1. merge the pair with the largest similairty
					//   2. merge muliple pairs in a time 
					//weight_i := (1 - sim_ij) * weight_i + sim_ij * weight_j
					caffe_gpu_axpby<Dtype>(K, Dtype(sim_ij), weight_data + j * K,
						Dtype(1 - sim_ij), weight_data + i * K);
					// NOTE: diff will be cleared in solver for all learnable params_
					// so it's not necessary to merge the difference here
					//refresh weight
					refresh_weight_cpu(j, weight, filler, K);
					merged_pos_index.insert(i);
					merged_pos_index.insert(j);
					LOG(INFO) << "weight_" << i << " and weight_" << j;
					break;
				}
			}//for (int j = i + 1; j < N; ++j)
		}//for (int i = 0; i < N; ++i)
		Dtype prop_merged = Dtype(merged_pos_index.size()) / Dtype(N) / Dtype(2);
		LOG(INFO) << prop_merged << " of the weights in \"" << name << "\" are merged(pos)";
		std::set<int> merged_neg_index;
		LOG(INFO) << "enhance negitive correlated weights:";
		for (int i = 0; i < N; ++i){
			if (merged_neg_index.count(i)){
				continue;
			}
			for (int j = i + 1; j < N; ++j){
				if (merged_neg_index.count(j)){
					continue;
				}
				const Dtype sim_ij = sim_data_cpu[i * N + j];
				if (sim_ij < -sim_th){
					// NOTE: other options: 
					//   1. merge the pair with the lowest similairty
					//   2. merge muliple pairs in a time 
					// weight_i := (1 - sim_ij) * weight_i + sim_ij * weight_j
					caffe_gpu_axpby<Dtype>(K, Dtype(sim_ij), weight_data + j * K,
						Dtype(1 - sim_ij), weight_data + i * K);
					// NOTE: diff will be cleared in solver for all learnable params_
					// so it's not necessary to merge the difference here
					// negative correlation we substract the correlated and enhance
					// its own weights instead of just randomly initialize one of them
					caffe_gpu_axpby<Dtype>(K, Dtype(sim_ij), weight_data + i * K,
						Dtype(1 - sim_ij), weight_data + j * K);
					// refresh weight
					// refresh_weight_cpu(j, weight, filler, K);
					merged_neg_index.insert(i);
					merged_neg_index.insert(j);
					LOG(INFO) << "weight_" << i << " and weight_" << j;
					break;
				}
			}//for (int j = i + 1; j < N; ++j)
		}//for (int i = 0; i < N; ++i)
		prop_merged = Dtype(merged_neg_index.size()) / Dtype(N) / Dtype(2);
		LOG(INFO) << prop_merged << " of the weights in \"" << name << "\" are enhanced(neg)";
	}

	template void merge_sim_weights_gpu<float>(Blob<float>* weight, Blob<float>* sim,
		const float prop, Filler<float>* filler, const int axis, string name,
		const bool hard);
	template void merge_sim_weights_gpu<double>(Blob<double>* weight, Blob<double>* sim,
		const double prop, Filler<double>* filler, const int axis, string name,
		const bool hard);
}// namespace caffe